#include "hip/hip_runtime.h"
/*
nvcc -o lorenz_cuda lorenz_cuda.cu -lGL -lGLU -lglut -lGLEW -lm
./lorenz_cuda
*/

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <GL/freeglut.h>
#include <GL/glut.h>
#include <GL/glu.h>
#include <GL/gl.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>

#define N 65536
#define NUM_BLOCKS 32
#define NUM_THREADS 2048

typedef struct point
{
    float x;
    float y;
    float z;
    float t;
} Point;

Point initial = {1.0, 1.0, 1.0, 0.0};

static GLfloat theta[] = {0.0, 0.0, 0.0};
GLint axis = 1;

__global__ void diff(Point *a)
{
    __shared__ float temp[4];
    __shared__ float sigma;
    __shared__ float rho;
    __shared__ float beta;
    __shared__ float dt;

    sigma = 10.0;
    rho = 28.0;
    beta = 8.0 / 3.0;
    dt = 0.01;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    switch (index)
    {
    case 0:
        temp[0] = sigma * (a->y - a->x);
        break;
    case 1:
        temp[1] = a->x * (rho - a->z) - a->y;
        break;
    case 2:
        temp[2] = a->x * a->y;
        break;
    case 3:
        temp[3] = beta * a->z;
        break;
    }

    // printf("%d\n", index);
    // printf("%f %f %f %f\n", temp[0], temp[1], temp[2], temp[3]);
    __syncthreads();

    if (index == 0)
    {
        printf("%f %f %f %f\n", temp[0], temp[1], temp[2], temp[3]);
        atomicAdd(&a->x, temp[0] * dt);
        atomicAdd(&a->y, temp[1] * dt);
        atomicAdd(&a->z, (temp[2] - temp[3]) * dt);
        atomicAdd(&a->t, dt);
    }
}

void draw(Point coord)
{
    glBegin(GL_POINTS);
    glColor3f(1, 1, 1);
    glVertex3f(coord.x, coord.y, coord.z);
    glEnd();
    glFlush();
    glutSwapBuffers();
}

void lorenzGenerator()
{
    Point *coord, *d_coord;
    coord = (Point *)malloc(sizeof(Point));
    hipMalloc((void **)&d_coord, sizeof(Point));
    *coord = initial;
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glLoadIdentity();
    glPointSize(1.0);
    for (unsigned long int i = 0; i < N; i++)
    {
        hipMemcpy(d_coord, coord, sizeof(Point), hipMemcpyHostToDevice);
        diff<<<NUM_BLOCKS, NUM_THREADS>>>(d_coord);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            printf("Error: %s\n", hipGetErrorString(err));

        hipMemcpy(coord, d_coord, sizeof(Point), hipMemcpyDeviceToHost);
        draw(*coord);
        break;
    }
    glutLeaveMainLoop();
    free(coord);
    hipFree(d_coord);
}

void myReshape(int w, int h)
{
    glRotatef(theta[0], 1.0, 0.0, 0.0);
    glRotatef(theta[1], 0.0, 1.0, 0.0);
    glRotatef(theta[2], 0.0, 0.0, 1.0);
    glViewport(0, 0, w, h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(-50.0, 50.0, -50.0, 50.0, -50.0, 50.0);
    glMatrixMode(GL_MODELVIEW);
}

void spinCube()
{
    if (theta[axis] > 360.0)
        theta[axis] -= 360.0;
    else if (theta[axis] < 0)
        theta[axis] += 360.0;
    glutPostRedisplay();
}

int main(int argc, char **argv)
{
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA | GLUT_DEPTH);
    glutInitWindowSize(1280, 720);
    glutCreateWindow("Lorenz");

    glutReshapeFunc(myReshape);
    glutIdleFunc(spinCube);
    glutDisplayFunc(lorenzGenerator);
    glEnable(GL_DEPTH_TEST);
    glutMainLoop();
}
